#include "src/operations/sigmoid.h"
#include "src/utils/matrix.h"

#include "hipblas.h"

#include "src/operations/hyperbolic_tan.h"
#include <math.h>
#include <stdio.h>
#include "src/utils/matrix.h"
#include "src/operations/add.h"
#include "src/utils/common.h"

#define N 4

void test_add();

int main(void) {
    //Matrix m = Matrix(N, N);
    //Matrix v = Matrix(N, 1);
    //Matrix res = Matrix(N, 1);

    float *d_m;
    float *m = (float *)malloc (N * N * sizeof (*m));
    float *d_v;
    float *v = (float *)malloc (N  * sizeof (*v));
    float *d_r;
    float *res = (float *)malloc (N * sizeof (*res));

    CHECK(hipMalloc((void **)&d_m, N*N*sizeof(*m)));
    CHECK(hipMalloc((void **)&d_v, N*1*sizeof(*v)));
    CHECK(hipMalloc((void **)&d_r, N*1*sizeof(*res)));

    float alpha = 1.0f;
    float beta = 1.0f;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    //m.allocate();
    //v.allocate();
    //res.allocate();

    srand (time(NULL));
    for(int r = 0; r < N; r++){
        for(int c = 0; c < N; c++)
            m[r*N+c] = r+c;
    }

    for(int r = 0; r < N; r++){
        for(int c = 0; c < 1; c++)
            v[r*1+c] = r+c;
    }
    //randimatrix(m, 5);
    //randimatrix(v, 5);
    printf("x = %d y = %d\n", N, N);
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*1+j]);
        printf("\n");
    }
    printf("\n");

    printf("x = %d y = %d\n", N, 1);
    for(int i = 0; i < N; i++){
        for(int j = 0; j < 1; j++)
            printf("%f ", v[i*1+j]);
        printf("\n");
    }
    printf("\n");

    //m.print_matrix();
    //v.print_matrix();

    //m.cpyHostToDev();
    //v.cpyHostToDev();
    CHECK(hipMemcpy(d_m, m, N*N*sizeof(*m), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_v, v, N*sizeof(*v), hipMemcpyHostToDevice));

    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, N,
            N, &alpha, d_m, N, d_v, 1, &beta, d_r, 1));
    hipDeviceSynchronize();

    CHECK(hipMemcpy(res, d_r, N*1*sizeof(float), hipMemcpyDeviceToHost));
    //res.cpyDevToHost();
    //res.print_matrix();

    printf("x = %d y = %d\n", N, 1);
    for(int i = 0; i < N; i++){
        for(int j = 0; j < 1; j++)
            printf("%f ", res[i*1+j]);
        printf("\n");
    }
    printf("\n");

    //m.destroy(); v.destroy();
    free(m);
    free(v);
    free(res);
    hipFree(d_m);
    hipFree(d_v);
    hipFree(d_r);

    hipblasDestroy(handle);

    printf("CUDA RESET");
    hipDeviceReset();
    return 0;
}

void test_add(){
    Matrix a = Matrix(N, N);
    Matrix b = Matrix(N, N);
    Matrix r;
    Matrix dX;

    Add x;

    srand (time(NULL));

    a.allocate();
    b.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            a[i*N+j] = (float) (rand() % 100);
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            b[i*N+j] = (float) (rand() % 100);
    }

    printf("A\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", a[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("B\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", b[i*N+j]);
        printf("\n");
    }
    printf("\n");

    a.cpyHostToDev();
    b.cpyHostToDev();

    r = x.forward(a, b);
    r.cpyDevToHost();

    dX = x.backward(a);
    dX.cpyDevToHost();


    printf("SOMMA\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", dX[i*N+j]);
        printf("\n");
    }
    printf("\n");

}

void test_sigmoid(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.forward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}
void test_tanh(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Tanh s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}