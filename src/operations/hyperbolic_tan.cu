#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-05-21.
//

#include "hyperbolic_tan.h"
#include "../utils/common.h"
#include <math.h>

__device__ float tanh_derivate(float x, float top_diff){
    return (1.0f - sqrt(x)) * top_diff;
}

__global__ void tanhForward(float* R, float* V, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        R[index] = tanh(V[index]); //TODO: parallelizzare tanh
}

__global__ void tanhBackward(float* dR, float* V, float *top_diff, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        dR[index] = tanh_derivate(V[index], top_diff[index]);
}

Matrix& Tanh::forward(Matrix &v) {
    this->V = v;
    R.allocate_size(v.getX(), v.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((V.getY() * V.getX() + TxB.x - 1) / TxB.x);
    tanhForward<<<num_blocks, TxB>>>(R.getDevData().get(), V.getDevData().get(), R.getX(), R.getY());

    return R;
}

Matrix& Tanh::backward(Matrix &top_diff) {
    dX.allocate_size(R.getX(), R.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((R.getY() * R.getX() + TxB.x - 1) / TxB.x);
    tanhBackward<<<num_blocks, TxB>>>(dX.getDevData().get(), R.getDevData().get(),
            top_diff.getDevData().get(), R.getX(), R.getY());

    return dX;
}