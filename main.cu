#include <stdio.h>
#include "src/operations/sigmoid.h"
#include "src/operations/matrix.h"

/*
La “Formula” di base
1. Setup dei dati su host (CPU-accessible memory)
2. Alloca memoria per i dati sulla GPU
3. Copia i dati da host a GPU
4. Alloca memoria per output su host
5. Alloca memoria per output su GPU
6. Lancia il kernel su GPU
7. Copia output da GPU a host
8. Cancella le memorie
 */

#define N 3

int main(void) {
    Matrix m = Matrix(N, N);
    m.allocate(true);

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) 8.0f;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    hipDeviceReset();
    return 0;
}