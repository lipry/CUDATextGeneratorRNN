#include "src/utils/matrix.h"
#include "src/operations/sigmoid.h"
#include "hipblas.h"
#include "src/operations/hyperbolic_tan.h"
#include <stdio.h>
#include "src/operations/add.h"
#include "src/utils/common.h"
#include "src/operations/prodmatvect.h"
#include "src/RNN/RnnLayer.h"
#include "src/utils/cudamath.h"
#include <cmath>
#include "src/operations/outputlayer.h"
#include "src/RNN/RnnNetwork.h"
#include <chrono>
#include <iostream>
#include <vector>


#define N 7
#define Neurons 6

//istanze N = 5000, Neurons = 5000, un loop di backpropagation su python 600ms, con cuda 5 ms.cx

int main() {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    //Matrix x = Matrix(N, 1);
    /*Matrix h_prev = Matrix(Neurons, 1);
    Matrix U = Matrix(Neurons, N);
    Matrix W = Matrix(Neurons, Neurons);
    Matrix V = Matrix(N, Neurons);

    Matrix diffh = Matrix(Neurons, 1);
    Matrix dVproduct = Matrix(N, 1);*/

    /*x.allocate();
    x.oneHotEncoder(2);
    x.print_matrix();*/

    /*Matrix M = Matrix(4, 3);
    M.allocate();
    int count = 1;
    for(int r = 0; r<4; r++){
        for(int c = 0; c<3; c++){
            M[r*3+c] = count;
            count++;
        }
    }
    M.cpyHostToDev();
    M.print_matrix();



    Matrix dz = Matrix(3, 1);
    dz.allocate();
    dz[0] = 1;
    dz[1] = 2;
    dz[2] = 3;
    dz[3] = 4;
    dz.cpyHostToDev();*/

    //float alpha = 1.0f;
    //float beta = 0.0f;

    /*size_t m = M.getY();
    size_t n = M.getX();
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, m, n,
                             &alpha, M.getDevData().get(), m, x.getDevData().get(), 1, &beta, out.getDevData().get(), 1));*/

    /*ProdMatVect prd = ProdMatVect();
    Matrix mul = prd.forward(handle, M, x);
    prd.backward(handle, dz);

    Matrix D = prd.getdMatrix();
    Matrix v = prd.getdVector();

    mul.cpyDevToHost();
    cout << "Mul: " << mul << endl;


    D.cpyDevToHost();
    printf("D: \n");
    D.print_matrix();

    v.cpyDevToHost();
    printf("v: \n");
    v.print_matrix();*/
    std::vector<int> v{ 2, 3, 2, 3, 4, 5, 6, 6, 6};
    std::vector<int> y{ 1, 2, 3, 4, 5, 6, 6, 6, 6};
    RnnNetwork rnn = RnnNetwork(N, Neurons);
    //const vector<RnnLayer> &layers = rnn.forward_prop(handle, v);
    rnn.backprop_through_time(handle, v, y, 3);

    //for(int i = 0; i < layers.size(); i++){
    //    cout << layers[i] << endl;
    //}

    /*Matrix U = rnn.getU();
    Matrix W = rnn.getW();
    Matrix V = rnn.getV();


    U.cpyDevToHost();
    W.cpyDevToHost();
    V.cpyDevToHost();

    printf("U: \n");
    U.print_matrix();
    printf("W: \n");
    W.print_matrix();
    printf("V: \n");
    V.print_matrix();*/



    //x.allocate();

    /*h_prev.allocate();
    diffh.allocate();
    dVproduct.allocate();*/

    /*for(int i = 0; i < N; i++){
        x[i] = 0.0;
    }
    x[2] = 1.0;
    cout << x;*/

    /*for(int i = 0; i < Neurons; i++){
        diffh[i] = i+1;
    }

    float step = 0.3f/N;
    float val = 0.0f;
    for(int i = 0; i < N; i++){
        dVproduct[i] = val;
        val += step;
    }


    step = 1.0f/Neurons;
    val = 0.0f;
    for(int i = 0; i < Neurons; i++){
        h_prev[i] = val;
        val += step;
    }

    U.allocate();
    for(int r = 0; r<Neurons; r++){
        step = 1.0f/N;
        val = 0.0f;
        for(int c = 0; c<N; c++){
            U[r*N+c] = val;
            val += step;
        }
    }

    W.allocate();
    for(int r = 0; r<Neurons; r++){
        step = 0.5f/Neurons;
        val = 0.0f;
        for(int c = 0; c<Neurons; c++){
            W[r*Neurons+c] = val;
            val += step;
        }
    }

    V.allocate();
    for(int r = 0; r<N; r++){
        step = 0.2f/Neurons;
        val = 0.8f;
        for(int c = 0; c<Neurons; c++){
            V[r*Neurons+c] = val;
            val += step;
        }
    }

    /*printf("X: \n");
    x.print_matrix();
    printf("h_prev: \n");
    h_prev.print_matrix();
    printf("U: \n");
    U.print_matrix();
    printf("W: \n");
    W.print_matrix();
    printf("V: \n");
    V.print_matrix();

    printf("diffh: \n");
    diffh.print_matrix();
    printf("dVProd: \n");
    dVproduct.print_matrix();*/

    /*diffh.cpyHostToDev();
    dVproduct.cpyHostToDev();

    x.cpyHostToDev();
    h_prev.cpyHostToDev();
    U.cpyHostToDev();
    W.cpyHostToDev();
    V.cpyHostToDev();



    RnnLayer rnnlayer;
    rnnlayer = RnnLayer();
    auto t1 = std::chrono::high_resolution_clock::now();
    rnnlayer.forward(handle, x, h_prev, U, W, V);
    rnnlayer.backward(handle, x, h_prev, U, W, V, diffh, dVproduct);
    auto t2 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();

    cout << "time elapsed: "<< duration / 1 << endl;

    Matrix h = rnnlayer.getH();
    Matrix output = rnnlayer.getOutput();
    Matrix dx = rnnlayer.getDx();
    Matrix dW = rnnlayer.getDW();
    Matrix dV = rnnlayer.getDV();
    Matrix dU = rnnlayer.getDU();
    Matrix dh_prev = rnnlayer.getDhPrev();

    h.cpyDevToHost();
    output.cpyDevToHost();
    dx.cpyDevToHost();
    dW.cpyDevToHost();
    dV.cpyDevToHost();
    dU.cpyDevToHost();
    dh_prev.cpyDevToHost();*/

    /*printf("h: \n");
    h.print_matrix();
    printf("output: \n");
    output.print_matrix();
    printf("dx \n");
    dx.print_matrix();
    printf("dW \n");
    dW.print_matrix();
    printf("dV \n");
    dV.print_matrix();
    printf("dU \n");
    dU.print_matrix();
    printf("dh_prev \n");
    dh_prev.print_matrix();*/
    hipblasDestroy(handle);
    hipDeviceReset();
    return 0;
}

/*
void test_sigmoid(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.forward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}
void test_tanh(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Tanh s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}

 void test_add(){
    Matrix a = Matrix(N, N);
    Matrix b = Matrix(N, N);
    Matrix r;
    Matrix dX;

    Add x;

    srand (time(NULL));

    a.allocate();
    b.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            a[i*N+j] = (float) (rand() % 100);
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            b[i*N+j] = (float) (rand() % 100);
    }

    printf("M\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", a[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("V\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", b[i*N+j]);
        printf("\n");
    }
    printf("\n");

    a.cpyHostToDev();
    b.cpyHostToDev();

    r = x.forward(a, b);
    r.cpyDevToHost();

    dX = x.backward(a);
    dX.cpyDevToHost();


    printf("SOMMA\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", r[i*N+j]);
        printf("\n");
    }
    printf("\n");
 }*/