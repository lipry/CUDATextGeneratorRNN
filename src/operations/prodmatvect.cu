#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-05-31.
//

#include "hipblas.h"
#include "../utils/common.h"
#include "prodmatvect.h"

__global__ void outerProduct(float *Res, float *A, float *B, int N)
{
    int i, j;
    // Determine matrix element row i and column j.
    i = blockIdx.y*blockDim.y + threadIdx.y;
    j = blockIdx.x*blockDim.x + threadIdx.x;

    // Each thread computes its own matrix element.
    printf("%f * %f = %f\n",A[i], B[j], A[i]*B[j]);
    Res[i*N + j] = A[i]*B[j];
}

/*__global__ void vect_vect(float *Res, float *A, float *B, int R, int C){
    int ROW = blockDim.y*blockIdx.y + threadIdx.y;
    int COL = blockDim.x*blockIdx.x + threadIdx.x;

    if((ROW < R) && (COL < C)){
        Res[ROW * C + COL] = 8;
        printf("(%d, %d, %d): %f* %f = %f \n", ROW, COL, ROW * C + COL);
    }else{
        printf("PISELLO (%d,%d) (%d,%d)\n", ROW, R, COL,C);//  idx = %f, idy = %f, x = %f, y = %f\n", idx, idy, x, y);
    }
}*/

Matrix& ProdMatVect::forward(Matrix& w, Matrix& v){
    /*if (w.getY() != v.getX())
        throw std::invalid_argument( "Matrix and Vectors dimension are not valid" );
    if(v.getY() != 1)
        throw std::invalid_argument( "V not a vector (Y != 1)" );*/

    this->W = w;
    this->V = v;
    R.allocate_size(v.getX(), 1);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, W.getX(),
            W.getY(), &alpha, W.getDevData().get(), W.getX(),
            V.getDevData().get(), 1, &beta, R.getDevData().get(), 1));

    hipblasDestroy(handle);

    return R;
}

void ProdMatVect::backward(Matrix &top_diff) {
    this->dW.allocate_size(W.getX(), W.getY());
    this->dv.allocate_size(top_diff.getX(), top_diff.getY());

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));


    //OK
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, W.getX(),
                             W.getY(), &alpha, W.getDevData().get(), W.getX(),
                             top_diff.getDevData().get(), 1, &beta, dv.getDevData().get(), 1));

    hipblasDestroy(handle);

    dim3 TxB(2, 2);
    dim3 num_blocks(dW.getY()/TxB.x, dW.getX()/TxB.y);
    outerProduct<<<num_blocks, TxB>>>(dW.getDevData().get(), V.getDevData().get(),
            top_diff.getDevData().get(), W.getX());

    /*CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, V.getX(),
                             V.getY(), &alpha, V.getDevData().get(), V.getX(),
                            top_diff.getDevData().get(), 1, &beta, dW.getDevData().get(), 1));
    printf("V.getX = %d\n", V.getX());
    printf("V.getY = %d\n", V.getY());
    printf("top_diff.getX = %d\n", top_diff.getX());
    printf("top_diff.getY = %d\n", top_diff.getY());
    printf("V\n");
    V.print_matrix();
    printf("top_diff\n");
    top_diff.print_matrix();
    CHECK_CUBLAS(hipblasSger(handle, dW.getX(), dW.getY(),
            &alpha, V.getDevData().get(), 1, top_diff.getDevData().get(), 1,
            dW.getDevData().get(), dW.getX()));

    hipblasDestroy(handle);

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((top_diff.getY() * top_diff.getX() + TxB.x - 1) / TxB.x);
    elemwise_prod_vect<<<num_blocks, TxB>>>(dW.getDevData().get(), V.getDevData().get(),
                            top_diff.getDevData().get(), top_diff.getX(),
                            top_diff.getY());*/
}

Matrix& ProdMatVect::getdW() {
    return this->dW;
}

Matrix& ProdMatVect::getdv() {
    return this->dv;
}