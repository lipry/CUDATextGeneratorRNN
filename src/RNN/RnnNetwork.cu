#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-09-09.
//
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hipblas.h"
#include <cmath>
#include <vector>
#include "../utils/cudamath.h"
#include "RnnNetwork.h"
#include "../utils/common.h"
#include "RnnLayer.h"
#include "../operations/outputlayer.h"


RnnNetwork::RnnNetwork(int input_dim, int hidden_dim) : input_dim(input_dim), hidden_dim(hidden_dim)
{
    init_weights();
}

void RnnNetwork::init_weights() {
    U.allocate_size(hidden_dim, input_dim);
    W.allocate_size(hidden_dim, hidden_dim);
    V.allocate_size(input_dim, hidden_dim);
    //U.load_rand(-sqrt(1.0f / input_dim), sqrt(1.0f / input_dim));
    //W.load_rand(-sqrt(1.0f / hidden_dim), sqrt(1.0f / hidden_dim));
    //V.load_rand(-sqrt(1.0f / hidden_dim), sqrt(1.0f / hidden_dim));
    cout << "ENTRO IN INIT WEIGGTTISSSSS DI MERDA" << endl;
    U.load_value(0.4);
    W.load_value(0.5);
    V.load_value(-0.6);

    U.cpyHostToDev();
    W.cpyHostToDev();
    V.cpyHostToDev();
}
/*
 *     def forward_prop(self, x):
        cells = []
        h_prev = np.zeros(self.hidden_dim)
        for elem in x:
            v = OneHotEncodingUtilities.one_hot_encoder(elem, self.input_dim)
            cell = RNNLayer()
            cell.forward_pass(v, h_prev, self.U, self.W, self.V)
            h_prev = cell.h
            cells.append(cell)
        return cells
 */
vector<RnnLayer> RnnNetwork::forward_prop(hipblasHandle_t handle, const std::vector<int> &x) {
    std::vector<RnnLayer> cells;
    //the allocation of shared pointer set values to 0.0 (I think)
    Matrix h_prev = Matrix(hidden_dim, 1);
    h_prev.allocate();
    //TODO: gestire meglio i trasferimenti in memoria
    //TODO: fare meglio l'init a zero (memset)
    for(int i=0;i<hidden_dim;i++){
        h_prev[i] = 0.0f;
    }
    h_prev.cpyHostToDev();

    Matrix v = Matrix(input_dim, 1);
    v.allocate();

    for (int elem : x){
        v.oneHotEncoder(elem);
        v.cpyHostToDev();
        RnnLayer cell = RnnLayer();
        cell.forward(handle, v, h_prev, this->U, this->W, this->V);
        h_prev = cell.getH();
        cells.push_back(cell);
    }

    /*for(std::vector<int>::size_type i = 0; i != cells.size(); i++) {
        cout << cells[i] << endl;
    }*/

    return cells;
}

/*
 *     def backprop_through_time(self, x, y, truncated=7):
        layers = self.forward_prop(x)
        T = len(layers)
        dU = np.zeros_like(self.U)
        dW = np.zeros_like(self.W)
        dV = np.zeros_like(self.V)

        output = OutputLayer()
        prev_ht = np.zeros(self.hidden_dim)
        diff_h = np.zeros(self.hidden_dim)
        for t in range(0, T):
            diff_Vprod = output.diff(layers[t].Vproduct, y[t])
            v = OneHotEncodingUtilities.one_hot_encoder(x[t], self.input_dim)
            _, dh_prev, dUt, dWt, dVt = layers[t].backward_pass(v, prev_ht, self.U, self.W, self.V, diff_h, diff_Vprod)
            prev_ht = layers[t].h
            diff_Vprod = np.zeros(self.input_dim)
            for i in range(t-1, max(t-1-truncated, -1), -1):
                v = OneHotEncodingUtilities.one_hot_encoder(x[i], self.input_dim)
                prev_hi = layers[i].h if i != 0 else np.zeros(self.hidden_dim)
                _, dh_prev, dUi, dWi, dVi = layers[i].backward_pass(v, prev_hi, self.U, self.W, self.V, dh_prev, diff_Vprod)
                dUt += dUi
                dWt += dWi
            dU += dUt
            dW += dWt
            dV += dVt
        return np.array([dU, dW, dV])
 */
void RnnNetwork::backprop_through_time(hipblasHandle_t handle, const std::vector<int> &x, const std::vector<int> &y,
                                       int truncated) {
    std::vector<RnnLayer> layers = this->forward_prop(handle, x);
    //u.cpyDevToHost();
    //cout << u << endl;
    // porcodio
    Matrix dU = Matrix(hidden_dim, input_dim);
    dU.allocate();
    dU.init_with_zeroes();
    dU.cpyHostToDev();
    Matrix dW = Matrix(hidden_dim, hidden_dim);
    dW.allocate();
    dW.init_with_zeroes();
    dW.cpyHostToDev();
    Matrix dV = Matrix(input_dim, hidden_dim);
    dV.allocate();
    dV.init_with_zeroes();
    dV.cpyHostToDev();

    Matrix prev_ht = Matrix(hidden_dim, 1);
    prev_ht.allocate();
    prev_ht.init_with_zeroes();
    prev_ht.cpyHostToDev();

    Matrix prev_hi = Matrix(hidden_dim, 1);
    prev_hi.allocate();
    prev_hi.init_with_zeroes();
    prev_hi.cpyHostToDev();

    Matrix diff_h = Matrix(hidden_dim, 1);
    diff_h.allocate();
    diff_h.init_with_zeroes();
    diff_h.cpyHostToDev();

    Matrix v = Matrix(input_dim, 1);
    v.allocate();

    Matrix diff_Vprod = Matrix(input_dim, 1);
    diff_Vprod.allocate();

    OutputLayer output = OutputLayer();

    for(std::vector<int>::size_type t = 0; t < layers.size(); t++){
        //TODO: problema diff?
        diff_Vprod = output.diff(handle, layers[t].getOutput(), y[t]);
        v.oneHotEncoder(x[t]);
        v.cpyHostToDev();

        diff_Vprod.cpyDevToHost();
        cout << "diff_VProd" << endl;
        cout << diff_Vprod << endl;

        // TODO: NON FUNZIONA BACKWARD, risultati diversi da python
        layers[t].backward(handle, v, prev_ht, U, W, V, diff_h, diff_Vprod);
        Matrix dx = layers[t].getDx();
        Matrix dh_prev = layers[t].getDhPrev();
        Matrix dUt = layers[t].getDU();
        Matrix dWt = layers[t].getDW();
        Matrix dVt = layers[t].getDV();

        dx.cpyDevToHost();
        cout << "dx" << endl;
        cout << dx << endl;
        dh_prev.cpyDevToHost();
        cout << "dh_prev" << endl;
        cout << dh_prev << endl;
        dUt.cpyDevToHost();
        cout << "dUt" << endl;
        cout << dUt << endl;
        dWt.cpyDevToHost();
        cout << "dWt" << endl;
        cout << dWt << endl;
        dVt.cpyDevToHost();
        cout << "dVt" << endl;
        cout << dVt << endl;
        break;

        prev_ht = layers[t].getH();
        diff_Vprod.init_with_zeroes();


        for(int i = t-1; i < max(int(t)-1-truncated, -1); i--){
            v.oneHotEncoder(x[i]);
            v.cpyDevToHost();

            if (i!=0){
                prev_hi = layers[i].getH();
            }else{
                prev_hi.init_with_zeroes();
            }

            layers[i].backward(handle, v, prev_hi, this->U, this->W, this->V, dh_prev, diff_Vprod);
        }
    }




}

const Matrix &RnnNetwork::getU() const {
    return U;
}

const Matrix &RnnNetwork::getW() const {
    return W;
}

const Matrix &RnnNetwork::getV() const {
    return V;
}
