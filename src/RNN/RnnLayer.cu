#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-06-14.
//
/*
 * class RNNLayer:
    def __init__(self):
        self.Uproduct = None
        self.Wproduct = None
        self.UWsum = None
        self.h = None
        self.Vproduct = None

    def forward_pass(self, x, h_prev, U, M, V):
        # calculating the state function h
        self.Uproduct = mul.forward_pass(U, x)
        self.Wproduct = mul.forward_pass(M, h_prev)
        self.UWsum = add.forward_pass(self.Uproduct, self.Wproduct)
        self.h = tanh.forward_pass(self.UWsum)

        self.Vproduct = mul.forward_pass(V, self.h)

    def backward_pass(self, x, h_prev, U, M, V, diffh, dVproduct):
        self.forward_pass(x, h_prev, U, M, V)
        dV, dhv = mul.backward_pass(V, self.h, dVproduct)
        dh = diffh + dhv
        dUWsum = tanh.backward_pass(self.UWsum, dh)
        dUproduct, dWproduct = add.backward_pass(self.Uproduct, self.Wproduct, dUWsum)
        dU, dx = mul.backward_pass(U, x, dUproduct)
        dM, dh_prev = mul.backward_pass(M, h_prev, dWproduct)
        return dx, dh_prev, dU, dM, dV
 */


#include "RnnLayer.h"
#include "../utils/common.h"
#include "../utils/cudamath.h"


void RnnLayer::forward(hipblasHandle_t handle, Matrix &x, Matrix &h_prev, Matrix &U, Matrix &W, Matrix &V){
    Matrix Uprod = Uproduct.forward(handle, U, x);
    Matrix Wprod = Wproduct.forward(handle, W, h_prev);
    Matrix UWs = UWsum.forward(Uprod, Wprod);
    this->h = ht.forward(UWs);
    this->output = Vhproduct.forward(handle, V, h);
}

void RnnLayer::backward(hipblasHandle_t handle, Matrix &x, Matrix &h_prev, Matrix &U,
        Matrix &W, Matrix &V, Matrix &diffh, Matrix &dVproduct){

    this->forward(handle, x, h_prev, U, W, V);
    Vhproduct.backward(handle, dVproduct);
    this->dV = Vhproduct.getdMatrix();
    Matrix dhv = Vhproduct.getdVector();

    Matrix dh;
    dh.allocate_size(dhv.getX(), dhv.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((dhv.getY() * dhv.getX() + TxB.x - 1) / TxB.x);
    add_vect<<<num_blocks, TxB>>>(dh.getDevData().get(), dhv.getDevData().get(),
            diffh.getDevData().get(), dhv.getX(), diffh.getY());

    Matrix dUwsum = ht.backward(dh);
    Matrix dUWproduct = UWsum.backward(dUwsum);
    Uproduct.backward(handle, dUWproduct);


    this->dx = Uproduct.getdVector();
    this->dU = Uproduct.getdMatrix();
    Wproduct.backward(handle, dUWproduct);
    this->dh_prev = Wproduct.getdVector();
    this->dW = Wproduct.getdMatrix();
}

const Matrix &RnnLayer::getH() const {
    return h;
}

const Matrix &RnnLayer::getOutput() const {
    return output;
}

const Matrix &RnnLayer::getDx() const {
    return dx;
}

const Matrix &RnnLayer::getDhPrev() const {
    return dh_prev;
}

const Matrix &RnnLayer::getDU() const {
    return dU;
}

const Matrix &RnnLayer::getDW() const {
    return dW;
}

const Matrix &RnnLayer::getDV() const {
    return dV;
}

const ProdMatVect &RnnLayer::getUproduct() const {
    return Uproduct;
}

const ProdMatVect &RnnLayer::getWproduct() const {
    return Wproduct;
}

const Add &RnnLayer::getUWsum() const {
    return UWsum;
}

const Tanh &RnnLayer::getHt() const {
    return ht;
}

const ProdMatVect &RnnLayer::getVhproduct() const {
    return Vhproduct;
}

std::ostream& operator<<(std::ostream &strm, const RnnLayer &cell) {
    cell.getH().cpyDevToHost();
    strm << "h: " << cell.getH() << endl;
    cell.getOutput().cpyDevToHost();
    strm << "output: " << cell.getOutput() << endl;
    cell.getDx().cpyDevToHost();
    strm << "dx: " << cell.getDx()<< endl;
    cell.getDhPrev().cpyDevToHost();
    strm << "dh_prev: " << cell.getDhPrev() << endl;
    cell.getDU().cpyDevToHost();
    strm << "dU: " << cell.getDU() << endl;
    cell.getDW().cpyDevToHost();
    strm << "dW: " << cell.getDW() << endl;
    cell.getDV().cpyDevToHost();
    strm << "dV: " << cell.getDV() << endl;
    return strm;
}