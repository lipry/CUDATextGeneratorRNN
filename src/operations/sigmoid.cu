#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-05-14.
//

#include <math.h>
#include "sigmoid.h"
#include <stdlib.h>
#include <stdio.h>

__device__ float sigmoid(float x){
    return 1.0f / (1 + exp(-x));
}

__global__ void sigmoidForward(float* R, float* V, int xdim){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < xdim)
        R[index] = sigmoid(V[index]);
}

void Sigmoid::forward(){
    int N = 12;
    float* A;
    float* B;
    float* d_A;
    float* d_B;
    int nBytes = N*sizeof(float);

    A = (float*)malloc(nBytes);
    B = (float*)malloc(nBytes);

    hipMalloc((void **)&d_A, nBytes);
    hipMalloc((void **)&d_B, nBytes);

    for(int i = 0; i < N; i++){
        B[i] = (float) i +1;
    }

    for(int i = 0; i < N; i++){
        printf("%f ", B[i]);
    }
    printf("\n");

    hipMemcpy(d_B, B, nBytes, hipMemcpyHostToDevice);

    sigmoidForward<<<N/3, 3>>>(d_A, d_B, N);

    hipMemcpy(A, d_A, nBytes, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        printf("%f ", A[i]);
    }

    free(A);
    free(B);

    hipFree(d_A);
    hipFree(d_B);

}
