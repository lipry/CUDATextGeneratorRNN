#include <stdio.h>
#include "src/operations/sigmoid.h"
#include "src/operations/matrix.h"

/*
La “Formula” di base
1. Setup dei dati su host (CPU-accessible memory)
2. Alloca memoria per i dati sulla GPU
3. Copia i dati da host a GPU
4. Alloca memoria per output su host
5. Alloca memoria per output su GPU
6. Lancia il kernel su GPU
7. Copia output da GPU a host
8. Cancella le memorie
 */

#define N 4

int main(void) {
    Matrix m = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    /*for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");*/

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    hipDeviceReset();
    return 0;
}