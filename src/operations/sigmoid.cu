#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-05-14.
//

#include <math.h>
#include "sigmoid.h"
#include "matrix.h"
#include "../utils/common.h"
#include <stdlib.h>
#include <stdio.h>

__device__ float sigmoid(float x){
    return 1.0f / (1 + exp(-x));
}

__global__ void sigmoidForward(float* R, float* V, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        R[index] = sigmoid(V[index]);
}

Matrix& Sigmoid::forward(Matrix &V){
    this->V = V;
    R.allocate_size(V.getX(), V.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((V.getY() * V.getX() + TxB.x - 1) / TxB.x);
    sigmoidForward<<<num_blocks, TxB>>>(R.getDevData().get(), V.getDevData().get(), R.getX(), R.getY());

    return R;

}
