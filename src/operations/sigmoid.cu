#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-05-14.
//

#include <math.h>
#include "sigmoid.h"
#include "../utils/matrix.h"
#include "../utils/common.h"
#include <stdlib.h>
#include <stdio.h>

__device__ float sigmoid(float x){
    return 1.0f / (1 + exp(-x)); //TODO: parallelizzare exp
}

__device__ float sigmoid_derivate(float x, float top_diff){
    return (1.0f - x)* x * top_diff;
}

__global__ void sigmoidForward(float* R, float* V, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        R[index] = sigmoid(V[index]);
}

__global__ void sigmoidBackward(float* dR, float* V, float *top_diff, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        dR[index] = sigmoid_derivate(V[index], top_diff[index]);
}

Matrix& Sigmoid::forward(Matrix &v){
    this->V = v;
    R.allocate_size(v.getX(), v.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((V.getY() * V.getX() + TxB.x - 1) / TxB.x);
    sigmoidForward<<<num_blocks, TxB>>>(R.getDevData(), V.getDevData(), R.getX(), R.getY());

    return R;

}

Matrix& Sigmoid::backward(Matrix &top_diff) {
    dX.allocate_size(R.getX(), R.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((R.getY() * R.getX() + TxB.x - 1) / TxB.x);
    sigmoidBackward<<<num_blocks, TxB>>>(dX.getDevData(), R.getDevData(),
            top_diff.getDevData(), R.getX(), R.getY());

    return dX;

}
