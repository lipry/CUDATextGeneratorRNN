#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-08-29.
//
#include "../utils/common.h"

__global__ void add_vect(float *R, float *A, float *B, int x, int y){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < x*y)
        R[idx] = A[idx] + B[idx]; // TODO: fare un altra funzione __device__?
}

__global__ void outerProduct(float *Res, float *A, float *B, int N)
{
    int i, j, x, y;
    x = threadIdx.x;
    y = threadIdx.y;
    i = blockIdx.y*blockDim.y + y;
    j = blockIdx.x*blockDim.x + x;

    __shared__ float shrA[BLOCK_SIZE];
    __shared__ float shrB[BLOCK_SIZE];

    if (x == 0)
        shrA[y] = A[i];
    __syncthreads();

    if(y == 0)
        shrB[x] = B[j];
    __syncthreads();

    Res[i*N + j] = shrA[y]*shrB[x];

    //printf("----- outer [%d,%d],[%d,%d]: %f * %f = %f\n", x, y, i, j,shrA[y], shrB[x], Res[i*N + j]);
}

__device__ float tanh_derivate(float x, float top_diff){
    return (1.0f - x*x) * top_diff;
}

__global__ void tanhForward(float* R, float* V, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        R[index] = tanh(V[index]); //TODO: parallelizzare tanh
}

__global__ void tanhBackward(float* dR, float* V, float *top_diff, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        dR[index] = tanh_derivate(V[index], top_diff[index]);
}

__device__ float sigmoid(float x){
    return 1.0f / (1 + exp(-x)); //TODO: parallelizzare exp
}

__device__ float sigmoid_derivate(float x, float top_diff){
    return (1.0f - x)* x * top_diff;
}

__global__ void sigmoidForward(float* R, float* V, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        R[index] = sigmoid(V[index]);
}

__global__ void sigmoidBackward(float* dR, float* V, float *top_diff, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        dR[index] = sigmoid_derivate(V[index], top_diff[index]);
}