#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-05-31.
//

#include "hipblas.h"
#include "../utils/common.h"
#include "prodmatvect.h"
#include "../utils/cudamath.h"


Matrix& ProdMatVect::forward(hipblasHandle_t handle, Matrix& w, Matrix& v){
    /*if (w.getY() != v.getX())
        throw std::invalid_argument( "Matrix and Vectors dimension are not valid" );
    if(v.getY() != 1)
        throw std::invalid_argument( "V not a vector (Y != 1)" );*/

    this->M = w;
    this->V = v;
    R.allocate_size(w.getX(), 1);

    float alpha = 1.0f;
    float beta = 0.0f;

    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_T, M.getY(),
            M.getX(), &alpha, M.getDevData().get(), M.getY(),
            V.getDevData().get(), 1, &beta, R.getDevData().get(), 1));

    return R;
}

void ProdMatVect::backward(hipblasHandle_t handle, Matrix &top_diff) {
    this->dM.allocate_size(top_diff.getX(), V.getX());
    this->dv.allocate_size(M.getY(), top_diff.getY());

    float alpha = 1.0f;
    float beta = 0.0f;
    //dv
    size_t m = M.getY();
    size_t n = M.getX();
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, m, n,
            &alpha, M.getDevData().get(), m, top_diff.getDevData().get(), 1, &beta, dv.getDevData().get(), 1));

    //dM
    dim3 TxB(BLOCK_SIZE, BLOCK_SIZE);
    dim3 num_blocks(ceil(float(dM.getY())/TxB.x), ceil(float(dM.getX())/TxB.y));
    outerProduct<<<num_blocks, TxB>>>(dM.getDevData().get(), top_diff.getDevData().get(),
            V.getDevData().get(), dM.getY());
}

Matrix& ProdMatVect::getdMatrix() {
    return this->dM;
}

Matrix& ProdMatVect::getdVector() {
    return this->dv;
}