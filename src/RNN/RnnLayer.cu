#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-06-14.
//
/*
 * class RNNLayer:
    def __init__(self):
        self.Uproduct = None
        self.Wproduct = None
        self.UWsum = None
        self.h = None
        self.Vproduct = None

    def forward_pass(self, x, h_prev, U, W, V):
        # calculating the state function h
        self.Uproduct = mul.forward_pass(U, x)
        self.Wproduct = mul.forward_pass(W, h_prev)
        self.UWsum = add.forward_pass(self.Uproduct, self.Wproduct)
        self.h = tanh.forward_pass(self.UWsum)

        self.Vproduct = mul.forward_pass(V, self.h)

    def backward_pass(self, x, h_prev, U, W, V, diffh, dVproduct):
        self.forward_pass(x, h_prev, U, W, V)
        dV, dhv = mul.backward_pass(V, self.h, dVproduct)
        dh = diffh + dhv
        dUWsum = tanh.backward_pass(self.UWsum, dh)
        dUproduct, dWproduct = add.backward_pass(self.Uproduct, self.Wproduct, dUWsum)
        dU, dx = mul.backward_pass(U, x, dUproduct)
        dW, dh_prev = mul.backward_pass(W, h_prev, dWproduct)
        return dx, dh_prev, dU, dW, dV
 */


#include "RnnLayer.h"

void RnnLayer::forward(Matrix &x, Matrix &h_prev, Matrix &U, Matrix &W, Matrix &V){
    /*
*       self.Uproduct = mul.forward_pass(U, x)
        self.Wproduct = mul.forward_pass(W, h_prev)
        self.UWsum = add.forward_pass(self.Uproduct, self.Wproduct)
        self.h = tanh.forward_pass(self.UWsum)

        self.Vproduct = mul.forward_pass(V, self.h)
     */
    Matrix Uprod = Uproduct.forward(U, x);
    Matrix Wprod = Wproduct.forward(W, h_prev);
    Matrix UWs = UWsum.forward(Uprod, Wprod);
    Matrix h = ht.forward(UWs);

    Matrix Vprod = Vhproduct.forward(V, h);

    Uprod.cpyDevToHost();
    Wprod.cpyDevToHost();
    UWs.cpyDevToHost();
    h.cpyDevToHost();
    Vprod.cpyDevToHost();
}