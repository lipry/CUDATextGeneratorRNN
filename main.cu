#include <stdio.h>
#include "src/operations/sigmoid.h"
#include "src/operations/matrix.h"

/*
La “Formula” di base
1. Setup dei dati su host (CPU-accessible memory)
2. Alloca memoria per i dati sulla GPU
3. Copia i dati da host a GPU
4. Alloca memoria per output su host
5. Alloca memoria per output su GPU
6. Lancia il kernel su GPU
7. Copia output da GPU a host
8. Cancella le memorie
 */

#define N 4

int main(void) {
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    hipDeviceReset();
    return 0;
}