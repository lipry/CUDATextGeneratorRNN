#include "hip/hip_runtime.h"
#include "src/operations/sigmoid.h"
#include "src/utils/matrix.h"

#include "hipblas.h"

#include "src/operations/hyperbolic_tan.h"
#include <math.h>
#include <stdio.h>
#include "src/utils/matrix.h"
#include "src/operations/add.h"
#include "src/utils/common.h"
#include "src/operations/prodmatvect.h"
#include "src/RNN/RnnLayer.h"
#include "src/utils/cudamath.h"

#define N 7
#define Neurons 6


int main(void) {
    Matrix x = Matrix(N, 1);
    Matrix h_prev = Matrix(Neurons, 1);
    Matrix U = Matrix(Neurons, N);
    Matrix W = Matrix(Neurons, Neurons);
    Matrix V = Matrix(N, Neurons);

    Matrix diffh = Matrix(Neurons, 1);
    Matrix dVproduct = Matrix(N, 1);

    x.allocate();
    h_prev.allocate();
    diffh.allocate();
    dVproduct.allocate();

    for(int i = 0; i < N; i++){
        x[i] = 0;
    }
    x[2] = 1;

    for(int i = 0; i < Neurons; i++){
        diffh[i] = i+1;
    }

    float step = 0.3f/N;
    float val = 0.0f;
    for(int i = 0; i < N; i++){
        dVproduct[i] = val;
        val += step;
    }


    step = 1.0f/Neurons;
    val = 0.0f;
    for(int i = 0; i < Neurons; i++){
        h_prev[i] = val;
        val += step;
    }

    U.allocate();
    for(int r = 0; r<Neurons; r++){
        step = 1.0f/N;
        val = 0.0f;
        for(int c = 0; c<N; c++){
            U[r*N+c] = val;
            val += step;
        }
    }

    W.allocate();
    for(int r = 0; r<Neurons; r++){
        step = 0.5f/Neurons;
        val = 0.0f;
        for(int c = 0; c<Neurons; c++){
            W[r*Neurons+c] = val;
            val += step;
        }
    }

    V.allocate();
    for(int r = 0; r<N; r++){
        step = 0.2f/Neurons;
        val = 0.8f;
        for(int c = 0; c<Neurons; c++){
            V[r*Neurons+c] = val;
            val += step;
        }
    }

    printf("X: \n");
    x.print_matrix();
    printf("h_prev: \n");
    h_prev.print_matrix();
    printf("U: \n");
    U.print_matrix();
    printf("W: \n");
    W.print_matrix();
    printf("V: \n");
    V.print_matrix();

    printf("diffh: \n");
    diffh.print_matrix();
    printf("dVProd: \n");
    dVproduct.print_matrix();

    diffh.cpyHostToDev();
    dVproduct.cpyHostToDev();

    x.cpyHostToDev();
    h_prev.cpyHostToDev();
    U.cpyHostToDev();
    W.cpyHostToDev();
    V.cpyHostToDev();

    RnnLayer rnnlayer = RnnLayer();
    rnnlayer.forward(x, h_prev, U, W, V);
    Matrix h = rnnlayer.getH();
    Matrix output = rnnlayer.getOutput();

    h.cpyDevToHost();
    output.cpyDevToHost();

    printf("h: \n");
    h.print_matrix();
    printf("output: \n");
    output.print_matrix();

    rnnlayer.backward(x, h_prev, U, W, V, diffh, dVproduct);

    /*Matrix dM;
    Matrix x = Matrix(N, 1);
    Matrix h_prev = Matrix(Neurons, 1);

    x.allocate();
    h_prev.allocate();
    dM.allocate_size(x.getX(), h_prev.getX());

    for(int i = 0; i < N; i++){
        x[i] = 0;
    }
    x[2] = 1;
    x[9] = 1;

    float step = 1.0f/Neurons;
    float val = 0.0f;
    for(int i = 0; i < Neurons; i++){
        h_prev[i] = val;
        val += step;
    }

    x.cpyHostToDev();
    h_prev.cpyHostToDev();
    printf("x: \n");
    x.print_matrix();
    printf("h_prev: \n");
    h_prev.print_matrix();


    dim3 TxB(BLOCK_SIZE, BLOCK_SIZE);
//dim3 num_blocks(dM.getY()/TxB.x, dM.getX()/TxB.y);
    dim3 num_blocks(ceil((dM.getY()/TxB.x)), ceil((dM.getX()/TxB.y)));

    printf("@@@ dim: %d,%d  %d,%d",   dM.getY(),TxB.x, dM.getX(),TxB.y);
    outerProduct<<<num_blocks, TxB>>>(dM.getDevData().get(), x.getDevData().get(),
            h_prev.getDevData().get(), dM.getY());

    dM.cpyDevToHost();
    dM.print_matrix();*/

    //printf("%f\n", ceil(7.0/2.0));


    hipDeviceReset();
    return 0;
}

/*
void test_sigmoid(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.forward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}
void test_tanh(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Tanh s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}

 void test_add(){
    Matrix a = Matrix(N, N);
    Matrix b = Matrix(N, N);
    Matrix r;
    Matrix dX;

    Add x;

    srand (time(NULL));

    a.allocate();
    b.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            a[i*N+j] = (float) (rand() % 100);
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            b[i*N+j] = (float) (rand() % 100);
    }

    printf("M\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", a[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("V\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", b[i*N+j]);
        printf("\n");
    }
    printf("\n");

    a.cpyHostToDev();
    b.cpyHostToDev();

    r = x.forward(a, b);
    r.cpyDevToHost();

    dX = x.backward(a);
    dX.cpyDevToHost();


    printf("SOMMA\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", r[i*N+j]);
        printf("\n");
    }
    printf("\n");
 }*/