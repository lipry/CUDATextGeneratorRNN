#include "src/operations/sigmoid.h"
#include "src/utils/matrix.h"

#include "hipblas.h"

#include "src/operations/hyperbolic_tan.h"
#include <math.h>
#include <stdio.h>
#include "src/utils/matrix.h"
#include "src/operations/add.h"
#include "src/utils/common.h"

#define N 4

void test_add();

int main(void) {
    Matrix m = Matrix(N, N);
    Matrix v = Matrix(N, 1);
    Matrix res = Matrix(N, 1);
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    m.allocate();
    v.allocate();
    res.allocate();

    randimatrix(m, 5);
    randimatrix(v, 5);

    m.print_matrix();
    v.print_matrix();

    m.cpyHostToDevCublas();
    v.cpyHostToDevCublas();


    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, m.getX(),
            m.getY(), &alpha, m.getDevData().get(), m.getX(), v.getDevData().get(), 1, &beta, res.getDevData().get(), 1));


    res.cpyDevToHostCublas();
    res.print_matrix();

    hipblasDestroy(handle);

    hipDeviceReset();
    return 0;
}

void test_add(){
    Matrix a = Matrix(N, N);
    Matrix b = Matrix(N, N);
    Matrix r;
    Matrix dX;

    Add x;

    srand (time(NULL));

    a.allocate();
    b.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            a[i*N+j] = (float) (rand() % 100);
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            b[i*N+j] = (float) (rand() % 100);
    }

    printf("A\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", a[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("B\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", b[i*N+j]);
        printf("\n");
    }
    printf("\n");

    a.cpyHostToDev();
    b.cpyHostToDev();

    r = x.forward(a, b);
    r.cpyDevToHost();

    dX = x.backward(a);
    dX.cpyDevToHost();


    printf("SOMMA\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", dX[i*N+j]);
        printf("\n");
    }
    printf("\n");

}

void test_sigmoid(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.forward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}
void test_tanh(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Tanh s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}