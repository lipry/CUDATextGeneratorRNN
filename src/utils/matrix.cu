//
// Created by Fabio Lipreri on 2019-05-14.
//
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../utils/cudamath.h"
#include "hipblas.h"
#include "matrix.h"
#include "common.h"
#include <bits/stdc++.h>

Matrix::Matrix(size_t x, size_t y) : x(x), y(y), host_alloc(false), dev_alloc(false),
dev_data(nullptr), host_data(nullptr)
{}

void Matrix::allocHostMemory() {
    if(!host_alloc){
        host_data = std::shared_ptr<float>(new float[x*y], [&](float *p) {delete[] p; });
        if(host_data)
            host_alloc = true;
    }
}

void Matrix::allocDevMemory() {
    if(!dev_alloc){
        float* tmp = nullptr;
        CHECK(hipMalloc((void **)&tmp, x*y*sizeof(float)));
        dev_data = std::shared_ptr<float>(tmp, [&](float *p){ hipFree(p); });
        if(dev_data)
            dev_alloc = true;
    }
}

void Matrix::allocate_size(size_t x, size_t y) {
    if(!dev_alloc && !host_alloc) {
        this->x = x;
        this->y = y;
        allocHostMemory();
        allocDevMemory();
    }
}

void Matrix::allocate() {
    if(!dev_alloc && !host_alloc) {
        allocHostMemory();
        allocDevMemory();
    }
}

bool Matrix::isDevAlloc(){
    return dev_alloc;
}

bool Matrix::isHostAlloc(){
    return host_alloc;
}

void Matrix::cpyHostToDev() {
    if(dev_alloc && host_alloc) {
        CHECK(hipMemcpy(dev_data.get(), host_data.get(), x * y * sizeof(float), hipMemcpyHostToDevice));
    }
}

void Matrix::cpyDevToHost() {
    if(dev_alloc && host_alloc)
        CHECK(hipMemcpy(host_data.get(), dev_data.get(), x*y*sizeof(float), hipMemcpyDeviceToHost));
}

void Matrix::cpyHostToDevCublas(){
    if(dev_alloc && host_alloc){
        //m = x, n = y
        if(getY() == 1) {
            printf("entro primo");
            CHECK_CUBLAS(hipblasSetVector(x, sizeof(float), host_data.get(), 1, dev_data.get(), 1));
        }else if(getX() == 1){
            printf("entro secondo");
            CHECK_CUBLAS(hipblasSetVector(y, sizeof(float), host_data.get(), 1, dev_data.get(), 1));
        }else {
            CHECK_CUBLAS(hipblasSetMatrix(x, y, sizeof(float), host_data.get(), x, dev_data.get(), y));
        }
    }

}

void Matrix::cpyDevToHostCublas(){
    if(dev_alloc && host_alloc){
        //m = x, n = y
        if(getY() == 1) {
            CHECK_CUBLAS(hipblasGetVector(x, sizeof(float), dev_data.get(), 1, host_data.get(), 1));
        }else if(getX() == 1){
            CHECK_CUBLAS(hipblasGetVector(y, sizeof(float), dev_data.get(), 1, host_data.get(), 1));
        }else{
            CHECK_CUBLAS(hipblasGetMatrix(x, y, sizeof(float), dev_data.get(), x, host_data.get(), y));
        }
    }
}

size_t Matrix::getX() const {
    return x;
}

size_t Matrix::getY() const {
    return y;
}

void Matrix::matrix_like(float number, Matrix &mat){
    allocate_size(mat.getY(), mat.getX());
    for(int i = 0; i<getX()*getY(); i++){
        this->host_data.get()[i] = number;
    }
}

bool Matrix::isVector(){
    // return getX() == 1; ???
    return getY() == 1 || getX() == 1;
}

void Matrix::print_matrix() {
    // TODO: column-major
    printf("x = %d y = %d\n", x, y);
    for(int i = 0; i < x; i++){
        for(int j = 0; j < y; j++)
            printf("%f ", host_data.get()[i*y+j]);
        printf("\n");
    }
    printf("\n");
}

void Matrix::load_rand(float lower, float higher) {
    hiprandState_t* states;
    int N = getX()*getY();
    /* allocate space on the GPU for the random states */
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((N + TxB.x - 1) / TxB.x);
    init_randoms<<<num_blocks, TxB>>>(time(0), states);
    randoms<<<num_blocks, TxB>>>(states, getDevData().get(), lower, higher);
}

const std::shared_ptr<float> &Matrix::getHostData() const {
    return host_data;
}

const std::shared_ptr<float> &Matrix::getDevData() const {
    return dev_data;
}

float& Matrix::operator[](const int index) {
    return host_data.get()[index];
}

const float& Matrix::operator[](const int index) const {
    return host_data.get()[index];
}
