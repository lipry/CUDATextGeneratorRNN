
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU (void) {
    int a = 0;
    printf("Hello from GPU!\n");
    printf("Max è scemo");
}

int main(void) {
    // hello from GPU
    int b = 0;
    printf("Hello World from CPU!\n");
    helloFromGPU <<<1, 10>>>();
    hipDeviceReset();
    return 0;
}