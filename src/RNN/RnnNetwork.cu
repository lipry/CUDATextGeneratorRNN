#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-09-09.
//
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include "../utils/cudamath.h"
#include "RnnNetwork.h"
#include "../utils/common.h"


RnnNetwork::RnnNetwork(int input_dim, int hidden_dim) : input_dim(input_dim), hidden_dim(hidden_dim)
{
    U.allocate_size(hidden_dim, input_dim);
    W.allocate_size(hidden_dim, hidden_dim);
    V.allocate_size(input_dim, hidden_dim);
    init_weights();
}

void RnnNetwork::init_weights() {
    U.load_rand(-sqrt(1.0f / input_dim), sqrt(1.0f / input_dim));
    W.load_rand(-sqrt(1.0f / hidden_dim), sqrt(1.0f / hidden_dim));
    V.load_rand(-sqrt(1.0f / hidden_dim), sqrt(1.0f / hidden_dim));
}

const Matrix &RnnNetwork::getU() const {
    return U;
}

const Matrix &RnnNetwork::getW() const {
    return W;
}

const Matrix &RnnNetwork::getV() const {
    return V;
}
