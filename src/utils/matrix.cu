//
// Created by Fabio Lipreri on 2019-05-14.
//
#include <iostream>
#include "common.h"
#include "matrix.h"

Matrix::Matrix(size_t x, size_t y) : x(x), y(y), host_alloc(false), dev_alloc(false),
dev_data(nullptr), host_data(nullptr)
{}

void Matrix::allocHostMemory() {
    if(!host_alloc){
        host_data = std::shared_ptr<float>(new float[x*y], [&](float *p) {delete[] p; });
        if(host_data)
            host_alloc = true;
    }
}

void Matrix::allocDevMemory() {
    if(!dev_alloc){
        float* tmp = nullptr;
        CHECK(hipMalloc((void **)&tmp, x*y*sizeof(float)));
        dev_data = std::shared_ptr<float>(tmp, [&](float *p){ hipFree(p); });
        if(dev_data)
            dev_alloc = true;
    }
}

void Matrix::allocate_size(size_t x, size_t y) {
    if(!dev_alloc && !host_alloc) {
        this->x = x;
        this->y = y;
        allocHostMemory();
        allocDevMemory();
    }
}

void Matrix::allocate() {
    if(!dev_alloc && !host_alloc) {
        allocHostMemory();
        allocDevMemory();
    }
}


void Matrix::cpyHostToDev() {
    if(dev_alloc && host_alloc) {
        CHECK(hipMemcpy(dev_data.get(), host_data.get(), x * y * sizeof(float), hipMemcpyHostToDevice));
    }
}

void Matrix::cpyDevToHost() {
    if(dev_alloc && host_alloc)
        CHECK(hipMemcpy(host_data.get(), dev_data.get(), x*y*sizeof(float), hipMemcpyDeviceToHost));
}

size_t Matrix::getX() const {
    return x;
}

size_t Matrix::getY() const {
    return y;
}

void Matrix::print_matrix() {
    for(int i = 0; i < x; i++){
        for(int j = 0; j < y; j++)
            printf("%f ", host_data.get()[i*y+j]);
        printf("\n");
    }
    printf("\n");
}

const std::shared_ptr<float> &Matrix::getHostData() const {
    return host_data;
}

const std::shared_ptr<float> &Matrix::getDevData() const {
    return dev_data;
}

float& Matrix::operator[](const int index) {
    return host_data.get()[index];
}

const float& Matrix::operator[](const int index) const {
    return host_data.get()[index];
}
