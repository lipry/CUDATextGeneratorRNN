#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-09-05.
//

/*
 * class OutputLayer:
    def predict(self, x):
        exp = np.exp(x-np.max(x))
        return exp / exp.sum(axis=0)

    def loss(self, x, y):
        p = self.predict(x)
        return -np.log(p[y])

    def diff(self, x, y):
        p = self.predict(x)
        p[y] -= 1
        return p
 */


#include <cmath>
#include "outputlayer.h"
#include "hipblas.h"
#include "../utils/common.h"
#include "../utils/cudamath.h"

void OutputLayer::predict(hipblasHandle_t handle, const Matrix &x){
    float sum = 0.0f;
    float alpha;
    int maxindex = -1;

    if(!predictions.isDevAlloc()){
        predictions.allocate_size(x.getX(), x.getY());
    }

    CHECK_CUBLAS(hipblasIsamax(handle, x.getX(), x.getDevData().get(), 1, &maxindex));
    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((x.getY() * x.getX() + TxB.x - 1) / TxB.x);
    exp_predict<<<num_blocks, TxB>>>(predictions.getDevData().get(), x.getDevData().get(), x[maxindex-1], x.getX(), x.getY());
    CHECK_CUBLAS(hipblasSasum(handle, predictions.getX(), predictions.getDevData().get(), 1, &sum))
    alpha = 1/sum;
    CHECK_CUBLAS(hipblasSscal(handle, predictions.getX(), &alpha, predictions.getDevData().get(), 1))
}

float OutputLayer::loss(hipblasHandle_t handle, const Matrix &x, int y) {
    if(!predictions.isDevAlloc()){
       this->predict(handle, x);
       predictions.cpyDevToHost();
    }
    //predictions.print_matrix();
    return -1.0 * log(predictions[y]);
}

const Matrix& OutputLayer::diff(hipblasHandle_t handle, const Matrix &x, int y){
    if(!predictions.isDevAlloc()){
        this->predict(handle, x);
        predictions.cpyDevToHost();
    }
    predictions[y] -= 1;
    predictions.cpyHostToDev();
    return predictions;
}

const Matrix &OutputLayer::getPredictions() const {
    return predictions;
}
