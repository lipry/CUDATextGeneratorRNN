#include "src/operations/sigmoid.h"
#include "src/utils/matrix.h"

#include "hipblas.h"

#include "src/operations/hyperbolic_tan.h"
#include <math.h>
#include <stdio.h>
#include "src/utils/matrix.h"
#include "src/operations/add.h"
#include "src/utils/common.h"
#include "src/operations/prodmatvect.h"

#define N 4

void test_add();

int main(void) {
    /*hipblasHandle_t handle;
    int len_x = 4;
    int len_y = 6;
    float alpha = 1.0f;
    float * x  = (float *) malloc(len_x * sizeof(float));
    float * y  = (float *) malloc(len_y * sizeof(float));
    float * res = (float *) malloc(len_x*len_y);

    float * d_x;
    float * d_y;
    float * d_res;

    hipMalloc((void **) &d_x, sizeof(float) * len_x);
    hipMalloc((void **) &d_y, sizeof(float) * len_y);
    hipMalloc((void **) &d_res, sizeof(float) * len_x * len_y);

    int ctr = 0;
    for(int r = 0; r < len_x; r++){ //vettore colonna
            x[r] =  5;//ctr++
    }

    ctr = 0;
    for (int c=0; c < len_y; c++) {
        y[c] = 7;//ctr++;
    }

    hipblasCreate(&handle);

    hipblasSetVector(len_x, sizeof(float), x, 1, d_x, 1);
    hipblasSetVector(len_y, sizeof(float), y, 1, d_y, 1);

    CHECK_CUBLAS(hipblasSger(handle, len_x, len_y,
                            &alpha, d_x, 1, d_y, 1,
                            d_res, len_x));

    hipblasGetMatrix(len_x, len_y, sizeof(float), d_res, len_x, res, len_x);

    printf("X\n");
    for(int r = 0; r < len_x; r++){ //vettore colonna
        printf("%f \n", x[r]);
    }
    printf("\n");

    printf("Y\n");
    for (int c=0; c < len_y; c++) {
        printf("%f ", y[c]); //ctr++;
    }
    printf("\n\n");


    printf("RES\n");
    for(int i = 0; i < len_x; i++){
        for(int j = 0; j < len_y; j++)
            printf("%f ", res[i*len_x+j]);
        printf("\n");
    }
    printf("\n");

    free(res);
    free(x);
    free(y);

    hipFree(d_res);
    hipFree(d_x);
    hipFree(d_y);*/
    Matrix m = Matrix(N, N);
    Matrix v = Matrix(N, 1);
    Matrix top_diff = Matrix(N, 1);

    Matrix res;
    Matrix porcodio;
    Matrix dv;

    ProdMatVect pmv = ProdMatVect();
    m.allocate();
    v.allocate();
    top_diff.allocate();

    randimatrix(m, 5);
    randimatrix(v, 5);
    randimatrix(top_diff, 5);

    printf("M\n");
    m.print_matrix();
    printf("V\n");
    v.print_matrix();
    printf("TOP DIFF\n");
    top_diff.print_matrix();

    m.cpyHostToDev();
    v.cpyHostToDev();
    top_diff.cpyHostToDev();

    res = pmv.forward(m, v);

    res.cpyDevToHost();

    printf("FORWARD\n");
    res.print_matrix();

    pmv.backward(top_diff);
    porcodio = pmv.getdW();
    dv = pmv.getdv();

    dv.cpyDevToHost();
    porcodio.cpyDevToHost();

    printf("DW\n");
    porcodio.print_matrix();
    printf("Dv\n");
    dv.print_matrix();
    //test_add();
    hipDeviceReset();
    return 0;
}

void test_add(){
    Matrix a = Matrix(N, N);
    Matrix b = Matrix(N, N);
    Matrix r;
    Matrix dX;

    Add x;

    srand (time(NULL));

    a.allocate();
    b.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            a[i*N+j] = (float) (rand() % 100);
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            b[i*N+j] = (float) (rand() % 100);
    }

    printf("W\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", a[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("V\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", b[i*N+j]);
        printf("\n");
    }
    printf("\n");

    a.cpyHostToDev();
    b.cpyHostToDev();

    r = x.forward(a, b);
    r.cpyDevToHost();

    dX = x.backward(a);
    dX.cpyDevToHost();


    printf("SOMMA\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", r[i*N+j]);
        printf("\n");
    }
    printf("\n");

}
/*
void test_sigmoid(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.forward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}
void test_tanh(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Tanh s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}*/