#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-05-31.
//

#include "hipblas.h"
#include "../utils/common.h"
#include "prodmatvect.h"

__global__ void outerProduct(float *Res, float *A, float *B, int N)
{
    int i, j, x, y;
    // Determine matrix element row i and column j.
    x = threadIdx.x;
    y = threadIdx.y;
    i = blockIdx.y*blockDim.y + y;
    j = blockIdx.x*blockDim.x + x;

    __shared__ float shrA[BLOCK_SIZE];
    __shared__ float shrB[BLOCK_SIZE];

    if (x == 0)
        shrA[y] = A[i];
    __syncthreads();

    if(y == 0)
        shrB[x] = B[j];
    __syncthreads();

    Res[i*N + j] = shrA[y]*shrB[x];
}

Matrix& ProdMatVect::forward(Matrix& w, Matrix& v){
    /*if (w.getY() != v.getX())
        throw std::invalid_argument( "Matrix and Vectors dimension are not valid" );
    if(v.getY() != 1)
        throw std::invalid_argument( "V not a vector (Y != 1)" );*/

    this->W = w;
    this->V = v;
    R.allocate_size(w.getX(), 1);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    //printf("W.getX(): %d\n", W.getX());
    //printf("W.getY(): %d\n", W.getY());
    //printf("W: ");
    //W.print_matrix();
    //printf("V: ");
    //V.print_matrix();


    //W.print_matrix();

    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_T, W.getY(),
            W.getX(), &alpha, W.getDevData().get(), W.getY(),
            V.getDevData().get(), 1, &beta, R.getDevData().get(), 1));

    hipblasDestroy(handle);

    return R;
}

void ProdMatVect::backward(Matrix &top_diff) {
    //TODO: CONTROLLARE TUTTO
    this->dW.allocate_size(W.getX(), W.getY());
    this->dv.allocate_size(top_diff.getX(), top_diff.getY());

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    //OK
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_T, W.getY(),
                             W.getX(), &alpha, W.getDevData().get(), W.getY(),
                             top_diff.getDevData().get(), 1, &beta, dv.getDevData().get(), 1));
    hipblasDestroy(handle);

    dim3 TxB(BLOCK_SIZE, BLOCK_SIZE);
    dim3 num_blocks(dW.getY()/TxB.x, dW.getX()/TxB.y);
    outerProduct<<<num_blocks, TxB>>>(dW.getDevData().get(), V.getDevData().get(),
            top_diff.getDevData().get(), W.getX());
}

Matrix& ProdMatVect::getdW() {
    return this->dW;
}

Matrix& ProdMatVect::getdv() {
    return this->dv;
}