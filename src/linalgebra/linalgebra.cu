//
// Created by Fabio Lipreri on 2019-05-06.
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#define INDEX(rows, cols, stride) (rows*stride + cols)
#define BDIMY 3
#define BDIMX 3

__global__ void add_vect(float *a, float *b, float *result, int n){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < n)
        result[idx] = a[idx] + b[idx];
}

__global__ void transpose_matrix(float *in, float *result, int nrows, int ncols){
    __shared__ float tile[BDIMY][BDIMX];

    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned int offset = INDEX(row, col, ncols);

    if (row < nrows && col < ncols)
        tile[threadIdx.y][threadIdx.x] = in[offset];

    __syncthreads();

    unsigned int bidx, irow, icol;
    bidx = threadIdx.y * blockDim.x + threadIdx.x;
    irow = bidx / blockDim.y;
    icol = bidx % blockDim.y;

    col = blockIdx.y * blockDim.y + icol;
    row = blockIdx.x * blockDim.x + irow;

    unsigned int transposed_offset = INDEX(row, col, nrows);

    if (row < ncols && col < nrows)
        result[transposed_offset] = tile[icol][irow];
}
