#include "src/operations/sigmoid.h"
#include "src/utils/matrix.h"

#include "hipblas.h"

#include "src/operations/hyperbolic_tan.h"
#include <math.h>
#include <stdio.h>
#include "src/utils/matrix.h"
#include "src/operations/add.h"
#include "src/utils/common.h"
#include "src/operations/prodmatvect.h"
#include "src/RNN/RnnLayer.h"

#define N 7
#define Neurons 6


int main(void) {
    Matrix x = Matrix(N, 1);
    Matrix h_prev = Matrix(Neurons, 1);
    Matrix U = Matrix(Neurons, N);
    Matrix W = Matrix(Neurons, Neurons);
    Matrix V = Matrix(N, Neurons);

    Matrix diffh = Matrix(Neurons, 1);
    Matrix dVproduct = Matrix(N, 1);

    x.allocate();
    h_prev.allocate();
    diffh.allocate();
    dVproduct.allocate();

    for(int i = 0; i < N; i++){
        x[i] = 0;
    }
    x[2] = 1;

    for(int i = 0; i < Neurons; i++){
        diffh[i] = i+1;
    }
    for(int i = 0; i < N; i++){
        dVproduct[i] = i+1;
    }



    for(int i = 0; i < Neurons; i++){
        h_prev[i] = 0;
    }

    U.allocate();
    for(int r = 0; r<Neurons; r++){
        for(int c = 0; c<N; c++){
            U[r*N+c] = c+1;
        }
    }

    W.allocate();
    for(int r = 0; r<Neurons; r++){
        for(int c = 0; c<Neurons; c++){
            W[r*Neurons+c] = c+1;
        }
    }

    V.allocate();
    for(int r = 0; r<N; r++){
        for(int c = 0; c<Neurons; c++){
            V[r*Neurons+c] = c+1;
        }
    }

    printf("X: \n");
    x.print_matrix();
    printf("h_prev: \n");
    h_prev.print_matrix();
    printf("U: \n");
    U.print_matrix();
    printf("W: \n");
    W.print_matrix();
    printf("V: \n");
    V.print_matrix();

    printf("diffh: \n");
    diffh.print_matrix();
    printf("dVProd: \n");
    dVproduct.print_matrix();

    diffh.cpyHostToDev();
    dVproduct.cpyHostToDev();

    x.cpyHostToDev();
    h_prev.cpyHostToDev();
    U.cpyHostToDev();
    W.cpyHostToDev();
    V.cpyHostToDev();

    RnnLayer rnnlayer = RnnLayer();
    //rnnlayer.forward(x, h_prev, U, W, V);
    rnnlayer.backward(x, h_prev, U, W, V, diffh, dVproduct);





    hipDeviceReset();
    return 0;
}

/*
void test_sigmoid(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.forward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}
void test_tanh(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Tanh s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}

 void test_add(){
    Matrix a = Matrix(N, N);
    Matrix b = Matrix(N, N);
    Matrix r;
    Matrix dX;

    Add x;

    srand (time(NULL));

    a.allocate();
    b.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            a[i*N+j] = (float) (rand() % 100);
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            b[i*N+j] = (float) (rand() % 100);
    }

    printf("W\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", a[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("V\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", b[i*N+j]);
        printf("\n");
    }
    printf("\n");

    a.cpyHostToDev();
    b.cpyHostToDev();

    r = x.forward(a, b);
    r.cpyDevToHost();

    dX = x.backward(a);
    dX.cpyDevToHost();


    printf("SOMMA\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", r[i*N+j]);
        printf("\n");
    }
    printf("\n");
 }*/