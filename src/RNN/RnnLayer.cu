#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-06-14.
//
/*
 * class RNNLayer:
    def __init__(self):
        self.Uproduct = None
        self.Wproduct = None
        self.UWsum = None
        self.h = None
        self.Vproduct = None

    def forward_pass(self, x, h_prev, U, M, V):
        # calculating the state function h
        self.Uproduct = mul.forward_pass(U, x)
        self.Wproduct = mul.forward_pass(M, h_prev)
        self.UWsum = add.forward_pass(self.Uproduct, self.Wproduct)
        self.h = tanh.forward_pass(self.UWsum)

        self.Vproduct = mul.forward_pass(V, self.h)

    def backward_pass(self, x, h_prev, U, M, V, diffh, dVproduct):
        self.forward_pass(x, h_prev, U, M, V)
        dV, dhv = mul.backward_pass(V, self.h, dVproduct)
        dh = diffh + dhv
        dUWsum = tanh.backward_pass(self.UWsum, dh)
        dUproduct, dWproduct = add.backward_pass(self.Uproduct, self.Wproduct, dUWsum)
        dU, dx = mul.backward_pass(U, x, dUproduct)
        dM, dh_prev = mul.backward_pass(M, h_prev, dWproduct)
        return dx, dh_prev, dU, dM, dV
 */


#include "RnnLayer.h"
#include "../utils/common.h"
#include "../utils/cudamath.h"

void RnnLayer::forward(Matrix &x, Matrix &h_prev, Matrix &U, Matrix &W, Matrix &V){
    /*
*       self.Uproduct = mul.forward_pass(U, x)
        self.Wproduct = mul.forward_pass(M, h_prev)
        self.UWsum = add.forward_pass(self.Uproduct, self.Wproduct)
        self.h = tanh.forward_pass(self.UWsum)

        self.Vproduct = mul.forward_pass(V, self.h)
     */
    Matrix Uprod = Uproduct.forward(U, x);
    printf("Uprod: \n");
    Uprod.cpyDevToHost();
    Uprod.print_matrix();
    Matrix Wprod = Wproduct.forward(W, h_prev);
    printf("Wprod: \n");
    Wprod.cpyDevToHost();
    Wprod.print_matrix();
    Matrix UWs = UWsum.forward(Uprod, Wprod);
    printf("UWs: \n");
    UWs.cpyDevToHost();
    UWs.print_matrix();
    this->h = ht.forward(UWs);
    this->output = Vhproduct.forward(V, h);
}

void RnnLayer::backward(Matrix &x, Matrix &h_prev, Matrix &U, Matrix &W, Matrix &V, Matrix &diffh, Matrix &dVproduct){
    /*self.forward_pass(x, h_prev, U, M, V)
    dV, dhv = mul.backward_pass(V, self.h, dVproduct)
    dh = diffh + dhv
    dUWsum = tanh.backward_pass(self.UWsum, dh)
    dUWproduct, dWproduct = add.backward_pass(self.Uproduct, self.Wproduct, dUWsum)
    dU, dx = mul.backward_pass(U, x, dUWproduct)
    dM, dh_prev = mul.backward_pass(M, h_prev, dWproduct)
    return dx, dh_prev, dU, dM, dV*/

    this->forward(x, h_prev, U, W, V);
    Vhproduct.backward(dVproduct);
    Matrix dV = Vhproduct.getdMatrix();
    Matrix dhv = Vhproduct.getdVector();

    dV.cpyDevToHost();
    dhv.cpyDevToHost();

    printf("dV: \n");
    dV.print_matrix();
    printf("dhv: \n");
    dhv.print_matrix();

    /*Matrix dh;
    dh.allocate_size(dhv.getX(), dhv.getY());

    // dh = diffh + dhv
    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((dhv.getY() * dhv.getX() + TxB.x - 1) / TxB.x);
    add_vect<<<num_blocks, TxB>>>(dh.getDevData().get(), dhv.getDevData().get(), diffh.getDevData().get(), dhv.getX(), diffh.getY());

    printf("dh: \n");
    dh.cpyDevToHost();
    dh.print_matrix();

    Matrix dUwsum = ht.backward(dh);

    printf("dUWsum: \n");
    dUwsum.cpyDevToHost();
    dUwsum.print_matrix();

    //dUWproduct == dWproduct nella derivata della somma
    Matrix dUWproduct = UWsum.backward(dUwsum);

    printf("------ dUWproduct: \n");
    dUWproduct.cpyDevToHost();
    dUWproduct.print_matrix();

    Wproduct.backward(dUWproduct);
    Matrix dh_prev = Wproduct.getdVector();
    Matrix dW = Wproduct.getdMatrix();

    printf("------ dh_prev: \n");
    dh_prev.cpyDevToHost();
    dh_prev.print_matrix();

    printf("------ dW: \n");
    dW.cpyDevToHost();
    dW.print_matrix();

    Uproduct.backward(dUWproduct);
    Matrix dx = Uproduct.getdVector();
    Matrix dU = Uproduct.getdMatrix();

    printf("------ dx: \n");
    dx.cpyDevToHost();
    dx.print_matrix();

    printf("------ dU: \n");
    dU.cpyDevToHost();
    dU.print_matrix();*/

}

const Matrix &RnnLayer::getH() const {
    return h;
}

const Matrix &RnnLayer::getOutput() const {
    return output;
}
