//
// Created by Fabio Lipreri on 2019-05-14.
//
#include <iostream>
#include "hipblas.h"
#include "matrix.h"
#include "common.h"

Matrix::Matrix(size_t x, size_t y) : x(x), y(y), host_alloc(false), dev_alloc(false),
dev_data(nullptr), host_data(nullptr)
{}

void Matrix::allocHostMemory() {
    if(!host_alloc){
        host_data = (float *)malloc (x * y * sizeof (*host_data));
        if(host_data)
            host_alloc = true;
    }
}

void Matrix::allocDevMemory() {
    if(!dev_alloc){
        CHECK(hipMalloc((void **)&dev_data, x*y*sizeof(*host_data)));
        if(dev_data)
            dev_alloc = true;
    }
}

void Matrix::allocate_size(size_t x, size_t y) {
    if(!dev_alloc && !host_alloc) {
        this->x = x;
        this->y = y;
        allocHostMemory();
        allocDevMemory();
    }
}

void Matrix::allocate() {
    if(!dev_alloc && !host_alloc) {
        allocHostMemory();
        allocDevMemory();
    }
}

void Matrix::destroy() {
    if(dev_alloc && host_alloc) {
        hipFree (dev_data);
        free(host_data);
        dev_data = nullptr;
        host_data = nullptr;
        printf("destroy");
    }
}

void Matrix::cpyHostToDev() {
    if(dev_alloc && host_alloc) {
        CHECK(hipMemcpy(dev_data, host_data, x * y * sizeof(float), hipMemcpyHostToDevice));
    }
}

void Matrix::cpyDevToHost() {
    if(dev_alloc && host_alloc)
        CHECK(hipMemcpy(host_data, dev_data, x*y*sizeof(float), hipMemcpyDeviceToHost));
}

void Matrix::cpyHostToDevCublas(){
    if(dev_alloc && host_alloc){
        //m = x, n = y
        if(isVector()) {
            CHECK_CUBLAS(hipblasSetVector(x, sizeof(float), host_data, 1, dev_data, 1));
        }else {
            CHECK_CUBLAS(hipblasSetMatrix(x, y, sizeof(float), host_data, x, dev_data, y));
        }
    }

}

void Matrix::cpyDevToHostCublas(){
    if(dev_alloc && host_alloc){
        //m = x, n = y
        if(isVector()){
            CHECK_CUBLAS(hipblasGetVector(x, sizeof(float), dev_data, 1, host_data, 1));
        }else{
            CHECK_CUBLAS(hipblasGetMatrix(x, y, sizeof(float), dev_data, x, host_data, y));
        }
    }
}

size_t Matrix::getX() const {
    return x;
}

size_t Matrix::getY() const {
    return y;
}

bool Matrix::isVector(){
    // return getX() == 1; ???
    return getY() == 1;
}

void Matrix::print_matrix() {
    // TODO: column-major
    printf("x = %d y = %d\n", x, y);
    for(int i = 0; i < x; i++){
        for(int j = 0; j < y; j++)
            printf("%f ", host_data[i*y+j]);
        printf("\n");
    }
    printf("\n");
}

float* Matrix::getHostData() const {
    return host_data;
}

float* Matrix::getDevData() const {
    return dev_data;
}

float& Matrix::operator[](const int index) {
    return host_data[index];
}

const float& Matrix::operator[](const int index) const {
    return host_data[index];
}
