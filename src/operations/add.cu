#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-05-14.
//

#include "add.h"
#include "../utils/common.h"

__global__ void add_vect(float *R, float *A, float *B, int x, int y){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < x*y)
        R[idx] = A[idx] + B[idx]; // TODO: fare un altra funzione __device__?
}

Matrix& Add::forward(Matrix& a, Matrix& b){
    this->A = a;
    this->B = b;
    R.allocate_size(a.getX(), a.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((A.getY() * A.getX() + TxB.x - 1) / TxB.x);
    add_vect<<<num_blocks, TxB>>>(R.getDevData(), A.getDevData(), B.getDevData(), A.getX(), A.getY());

    return R;
}

Matrix& Add::backward(Matrix& top_diff){
    this->dX = top_diff;
    return dX;
}