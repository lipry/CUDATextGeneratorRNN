//
// Created by Fabio Lipreri on 2019-05-14.
//


#include "add.h"
#include "hipblas.h"
#include "../utils/common.h"
#include "../utils/cudamath.h"

Matrix& Add::forward(Matrix& a, Matrix& b){
    this->A = a;
    this->B = b;
    R.allocate_size(a.getX(), a.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((A.getY() * A.getX() + TxB.x - 1) / TxB.x);
    add_vect<<<num_blocks, TxB>>>(R.getDevData().get(), A.getDevData().get(), B.getDevData().get(), A.getX(), A.getY());

    return R;
}

Matrix& Add::backward(Matrix& top_diff){
    this->dA = top_diff;
    return dA;
}