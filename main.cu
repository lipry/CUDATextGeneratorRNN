#include "src/operations/sigmoid.h"
#include "src/utils/matrix.h"

#include "hipblas.h"

#include "src/operations/hyperbolic_tan.h"
#include <math.h>
#include <stdio.h>
#include "src/utils/matrix.h"
#include "src/operations/add.h"
#include "src/utils/common.h"
#include "src/operations/prodmatvect.h"
#include "src/RNN/RnnLayer.h"
#include "src/utils/cudamath.h"
#include <chrono>
#include <iostream>


#define N 7
#define Neurons 6

//istanze N = 50000, Neurons = 5000, un loop di backpropagation su python 600ms, con cuda 5 ms.

int main(void) {

    Matrix x = Matrix(N, 1);
    Matrix h_prev = Matrix(Neurons, 1);
    Matrix U = Matrix(Neurons, N);
    Matrix W = Matrix(Neurons, Neurons);
    Matrix V = Matrix(N, Neurons);

    Matrix diffh = Matrix(Neurons, 1);
    Matrix dVproduct = Matrix(N, 1);

    x.allocate();
    h_prev.allocate();
    diffh.allocate();
    dVproduct.allocate();

    for(int i = 0; i < N; i++){
        x[i] = 0;
    }
    x[2] = 1;

    for(int i = 0; i < Neurons; i++){
        diffh[i] = i+1;
    }

    float step = 0.3f/N;
    float val = 0.0f;
    for(int i = 0; i < N; i++){
        dVproduct[i] = val;
        val += step;
    }


    step = 1.0f/Neurons;
    val = 0.0f;
    for(int i = 0; i < Neurons; i++){
        h_prev[i] = val;
        val += step;
    }

    U.allocate();
    for(int r = 0; r<Neurons; r++){
        step = 1.0f/N;
        val = 0.0f;
        for(int c = 0; c<N; c++){
            U[r*N+c] = val;
            val += step;
        }
    }

    W.allocate();
    for(int r = 0; r<Neurons; r++){
        step = 0.5f/Neurons;
        val = 0.0f;
        for(int c = 0; c<Neurons; c++){
            W[r*Neurons+c] = val;
            val += step;
        }
    }

    V.allocate();
    for(int r = 0; r<N; r++){
        step = 0.2f/Neurons;
        val = 0.8f;
        for(int c = 0; c<Neurons; c++){
            V[r*Neurons+c] = val;
            val += step;
        }
    }

    /*printf("X: \n");
    x.print_matrix();
    printf("h_prev: \n");
    h_prev.print_matrix();
    printf("U: \n");
    U.print_matrix();
    printf("W: \n");
    W.print_matrix();
    printf("V: \n");
    V.print_matrix();

    printf("diffh: \n");
    diffh.print_matrix();
    printf("dVProd: \n");
    dVproduct.print_matrix();

    diffh.cpyHostToDev();
    dVproduct.cpyHostToDev();*/

    x.cpyHostToDev();
    h_prev.cpyHostToDev();
    U.cpyHostToDev();
    W.cpyHostToDev();
    V.cpyHostToDev();

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    //Matrix h = rnnlayer.getH();
    //Matrix output = rnnlayer.getOutput();

    //h.cpyDevToHost();
    //output.cpyDevToHost();

    /*printf("h: \n");
    h.print_matrix();
    printf("output: \n");
    output.print_matrix();*/
    RnnLayer rnnlayer;
    auto t1 = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < 20; i++){
        rnnlayer = RnnLayer();
        //rnnlayer.forward(handle, x, h_prev, U, W, V);
        rnnlayer.backward(handle, x, h_prev, U, W, V, diffh, dVproduct);
    }
    auto t2 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();

    cout << "time elapsed: "<< duration / 20 << endl;

    Matrix dx = rnnlayer.getDx();
    Matrix dW = rnnlayer.getDW();
    Matrix dV = rnnlayer.getDV();
    Matrix dU = rnnlayer.getDU();
    Matrix dh_prev = rnnlayer.getDhPrev();

    dx.cpyDevToHost();
    dW.cpyDevToHost();
    dV.cpyDevToHost();
    dU.cpyDevToHost();
    dh_prev.cpyDevToHost();

    /*printf("dx \n");
    dx.print_matrix();
    printf("dW \n");
    dW.print_matrix();
    printf("dV \n");
    dV.print_matrix();
    printf("dU \n");
    dU.print_matrix();
    printf("dh_prev \n");
    dh_prev.print_matrix();*/
    hipblasDestroy(handle);
    hipDeviceReset();
    return 0;
}

/*
void test_sigmoid(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.forward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}
void test_tanh(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Tanh s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}

 void test_add(){
    Matrix a = Matrix(N, N);
    Matrix b = Matrix(N, N);
    Matrix r;
    Matrix dX;

    Add x;

    srand (time(NULL));

    a.allocate();
    b.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            a[i*N+j] = (float) (rand() % 100);
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            b[i*N+j] = (float) (rand() % 100);
    }

    printf("M\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", a[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("V\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", b[i*N+j]);
        printf("\n");
    }
    printf("\n");

    a.cpyHostToDev();
    b.cpyHostToDev();

    r = x.forward(a, b);
    r.cpyDevToHost();

    dX = x.backward(a);
    dX.cpyDevToHost();


    printf("SOMMA\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", r[i*N+j]);
        printf("\n");
    }
    printf("\n");
 }*/