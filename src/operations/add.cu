//
// Created by Fabio Lipreri on 2019-05-14.
//


#include "add.h"
#include "hipblas.h"
#include "../utils/common.h"
#include "../utils/cudamath.h"

Matrix& Add::forward(Matrix& a, Matrix& b){
    this->A = a;
    this->B = b;
    R.allocate_size(a.getX(), a.getY());

    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((A.getY() * A.getX() + TxB.x - 1) / TxB.x);
    add_vect<<<num_blocks, TxB>>>(R.getDevData().get(), A.getDevData().get(), B.getDevData().get(), A.getX(), A.getY());

    return R;
}

Matrix& Add::backward(Matrix& top_diff){
    this->dA = top_diff;
//    Matrix a_like;
//    Matrix b_like;
//    a_like.matrix_like(1.0f, A);
//    b_like.matrix_like(1.0f, B);
//    a_like.print_matrix();
//    b_like.print_matrix();
//    a_like.cpyHostToDev();
//    b_like.cpyHostToDev();
//
//    this->dA.allocate_size(a_like.getX(), 1);
//    this->dB.allocate_size(b_like.getX(), 1);
//
//    float alpha = 1.0f;
//    float beta = 0.0f;
//
//    hipblasHandle_t handle;
//    CHECK_CUBLAS(hipblasCreate(&handle));
//
//    size_t m = a_like.getY();
//    size_t n = a_like.getX();
//    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, m, n,
//                             &alpha, a_like.getDevData().get(), m, top_diff.getDevData().get(), 1,
//                             &beta, dA.getDevData().get(), 1));
//
//    hipblasDestroy(handle);

    return dA;
}