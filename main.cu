#include "src/operations/sigmoid.h"
#include "src/utils/matrix.h"

#include "hipblas.h"

#include "src/operations/hyperbolic_tan.h"
#include <math.h>
#include <stdio.h>
#include "src/utils/matrix.h"
#include "src/operations/add.h"
#include "src/utils/common.h"
#include "src/operations/prodmatvect.h"

#define N 4

void test_add();

int main(void) {
    Matrix m = Matrix(N, N);
    Matrix v = Matrix(N, 1);
    Matrix top_diff = Matrix(N, 1);

    Matrix res;
    Matrix dW;
    Matrix dv;

    ProdMatVect pmv = ProdMatVect();
    m.allocate();
    v.allocate();
    top_diff.allocate();

    randimatrix(m, 5);
    randimatrix(v, 5);
    randimatrix(top_diff, 5);

    printf("M\n");
    m.print_matrix();
    printf("V\n");
    v.print_matrix();
    printf("TOP DIFF\n");
    top_diff.print_matrix();

    m.cpyHostToDev();
    v.cpyHostToDev();
    top_diff.cpyHostToDev();

    res = pmv.forward(m, v);

    res.cpyDevToHost();

    printf("FORWARD\n");
    res.print_matrix();

    pmv.backward(top_diff);
    dW = pmv.getdW();
    dv = pmv.getdv();

    dv.cpyDevToHost();
    dW.cpyDevToHost();

    printf("DW\n");
    dW.print_matrix();
    printf("Dv\n");
    dv.print_matrix();
    //test_add();
    hipDeviceReset();
    return 0;
}

void test_add(){
    Matrix a = Matrix(N, N);
    Matrix b = Matrix(N, N);
    Matrix r;
    Matrix dX;

    Add x;

    srand (time(NULL));

    a.allocate();
    b.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            a[i*N+j] = (float) (rand() % 100);
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            b[i*N+j] = (float) (rand() % 100);
    }

    printf("W\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", a[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("V\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", b[i*N+j]);
        printf("\n");
    }
    printf("\n");

    a.cpyHostToDev();
    b.cpyHostToDev();

    r = x.forward(a, b);
    r.cpyDevToHost();

    dX = x.backward(a);
    dX.cpyDevToHost();


    printf("SOMMA\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", r[i*N+j]);
        printf("\n");
    }
    printf("\n");

}
/*
void test_sigmoid(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Sigmoid s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.forward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}
void test_tanh(){
    Matrix m = Matrix(N, N);
    Matrix top_diff = Matrix(N, N);
    //Matrix r;
    Tanh s;

    m.allocate();
    top_diff.allocate();

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            m[i*N+j] = (float) i+j;
    }

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            top_diff[i*N+j] = (float) i-j;
    }

    printf("ORIGINALE\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    printf("TOP DIFF\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", top_diff[i*N+j]);
        printf("\n");
    }
    printf("\n");

    m.cpyHostToDev();
    m = s.forward(m);
    m.cpyDevToHost();

    printf("DOPO FORWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");

    top_diff.cpyHostToDev();
    m = s.backward(top_diff);
    m.cpyDevToHost();

    printf("DOPO BACKWARD\n");
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            printf("%f ", m[i*N+j]);
        printf("\n");
    }
    printf("\n");
}*/